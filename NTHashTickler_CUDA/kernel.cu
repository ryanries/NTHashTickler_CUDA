#include "hip/hip_runtime.h"
// NTHashTickler_CUDA
// By Ryan Ries, 2014, ryanries09@gmail.com, myotherpcisacloud.com
// 

#include <stdio.h>
#include <conio.h>
#include <ctype.h>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <Windows.h>
#include <helper_cuda_drvapi.h>
#include <drvapi_error_string.h>
#include <>
#include <bcrypt.h>

int preferredDeviceNum = 0;

bool host_shouldStop = false;
__device__ __managed__ unsigned char randomBytes[32768];
__device__ __managed__ unsigned char inputHashBytes[16] = { 0 };
__device__ __managed__ int maxPasswordLength = 12;
__device__ __managed__ bool device_shouldStop = false;

// These are initialization values for the MD4 hash algorithm. See RFC 1320.
__constant__ uint32_t INIT_A = 0x67452301;
__constant__ uint32_t INIT_B = 0xefcdab89;
__constant__ uint32_t INIT_C = 0x98badcfe;
__constant__ uint32_t INIT_D = 0x10325476;
__constant__ uint32_t SQRT_2 = 0x5a827999;
__constant__ uint32_t SQRT_3 = 0x6ed9eba1;

// Common password characters.
__constant__ char validChars[] = { 0x20, 0x21, 0x22, 0x23, 0x24, 0x25, 0x26, 0x27, 0x28, 0x29, 0x2A, 0x2B, 0x2C, 0x2D, 0x2E, 0x2F,
                                   0x30, 0x31, 0x32, 0x33, 0x34, 0x35, 0x36, 0x37, 0x38, 0x39, 0x3A, 0x3B, 0x3C, 0x3D, 0x3E, 0x3F,
                                   0x40, 0x41, 0x42, 0x43, 0x44, 0x45, 0x46, 0x47, 0x48, 0x49, 0x4A, 0x4B, 0x4C, 0x4D, 0x4E, 0x4F,
                                   0x50, 0x51, 0x52, 0x53, 0x54, 0x55, 0x56, 0x57, 0x58, 0x59, 0x4A, 0x5B, 0x5C, 0x5D, 0x5E, 0x5F,
                                   0x60, 0x61, 0x62, 0x63, 0x64, 0x65, 0x66, 0x67, 0x68, 0x69, 0x6A, 0x6B, 0x6C, 0x6D, 0x6E, 0x6F,
                                   0x70, 0x71, 0x72, 0x73, 0x74, 0x75, 0x76, 0x77, 0x78, 0x79, 0x7A, 0x7B, 0x7C, 0x7D, 0x7E };

// Does string base begin with substring str?
bool startsWith(char * base, char * str) 
{
	return (strstr(base, str) - base) == 0;
}

// Is this string 32 characters long and only consists of hex chars?
bool IsMD4HashString(char * input)
{
	const char nibbles[] = { "0123456789abcdefABCDEF" };

	if (strlen(input) != 32)
		return false;	

	for (int x = 0; input[x]; x++)
	{
		bool isNibble = false;
		for (int y = 0; y < 16; y++)
		{
			if (nibbles[y] == input[x])
				isNibble = true;
		}
		if (isNibble == false)
			return false;
	}
	return true;
}

// Print some helpful text and exit the program.
void PrintHelpText(char * extraMessage)
{	
	printf("\nNTHashTickler_CUDA v1.0\n");
	printf("-----------------------\n");
	printf("Written by Ryan Ries, myotherpcisacloud.com\n\n");
	printf("Usage: C:\\> NTHashTickler_CUDA hash=d79e1c308aa5bbcdeea8ed63df412da9\n");
	printf("                               [maxlen=8]\n");
	printf("                               [preferredDevice=0]\n");
	printf("\nUses brute force to find a plain text input that generates an\n");
	printf("NT hash that matches the one supplied by the user. An NT (or NTLM)\n");
	printf("hash is the MD4 hash of the Unicode little endian plain text.\n");
	printf("Arguments are not case sensitive.\n\n");
	printf("The maxlen argument is optional and specifies the maximum password\n");
	printf("length for which to generate hashes. Default is 12.\n\n");
	printf("The preferredDevice argument specifies the NVIDIA CUDA device to use.\n");
	printf("Only a single CUDA device is supported at this time because I only had\n");
	printf("one to test with. Default is 0 (the first CUDA device found) but you\n");
	printf("can override that if your second or third device is faster.\n");
	printf("\n%s\n", extraMessage);
	exit(EXIT_FAILURE);
}

__device__ void NTHash(char * password, int length, uint32_t * output)
{
	uint32_t nt_buffer[16] = { 0 };
	int i = 0;
	
	//The length of key need to be <= 27
	for (; i<length / 2; i++)
		nt_buffer[i] = password[2 * i] | (password[2 * i + 1] << 16);

	// Padding
	if (length % 2 == 1)
		nt_buffer[i] = password[length - 1] | 0x800000;
	else
		nt_buffer[i] = 0x80;

	nt_buffer[14] = length << 4;

	uint32_t a = INIT_A;
	uint32_t b = INIT_B;
	uint32_t c = INIT_C;
	uint32_t d = INIT_D;

	/* Round 1 */
	a += (d ^ (b & (c ^ d))) + nt_buffer[0]; a = (a << 3) | (a >> 29);
	d += (c ^ (a & (b ^ c))) + nt_buffer[1]; d = (d << 7) | (d >> 25);
	c += (b ^ (d & (a ^ b))) + nt_buffer[2]; c = (c << 11) | (c >> 21);
	b += (a ^ (c & (d ^ a))) + nt_buffer[3]; b = (b << 19) | (b >> 13);

	a += (d ^ (b & (c ^ d))) + nt_buffer[4]; a = (a << 3) | (a >> 29);
	d += (c ^ (a & (b ^ c))) + nt_buffer[5]; d = (d << 7) | (d >> 25);
	c += (b ^ (d & (a ^ b))) + nt_buffer[6]; c = (c << 11) | (c >> 21);
	b += (a ^ (c & (d ^ a))) + nt_buffer[7]; b = (b << 19) | (b >> 13);

	a += (d ^ (b & (c ^ d))) + nt_buffer[8]; a = (a << 3) | (a >> 29);
	d += (c ^ (a & (b ^ c))) + nt_buffer[9]; d = (d << 7) | (d >> 25);
	c += (b ^ (d & (a ^ b))) + nt_buffer[10]; c = (c << 11) | (c >> 21);
	b += (a ^ (c & (d ^ a))) + nt_buffer[11]; b = (b << 19) | (b >> 13);

	a += (d ^ (b & (c ^ d))) + nt_buffer[12]; a = (a << 3) | (a >> 29);
	d += (c ^ (a & (b ^ c))) + nt_buffer[13]; d = (d << 7) | (d >> 25);
	c += (b ^ (d & (a ^ b))) + nt_buffer[14]; c = (c << 11) | (c >> 21);
	b += (a ^ (c & (d ^ a))) + nt_buffer[15]; b = (b << 19) | (b >> 13);

	/* Round 2 */
	a += ((b & (c | d)) | (c & d)) + nt_buffer[0] + SQRT_2; a = (a << 3) | (a >> 29);
	d += ((a & (b | c)) | (b & c)) + nt_buffer[4] + SQRT_2; d = (d << 5) | (d >> 27);
	c += ((d & (a | b)) | (a & b)) + nt_buffer[8] + SQRT_2; c = (c << 9) | (c >> 23);
	b += ((c & (d | a)) | (d & a)) + nt_buffer[12] + SQRT_2; b = (b << 13) | (b >> 19);

	a += ((b & (c | d)) | (c & d)) + nt_buffer[1] + SQRT_2; a = (a << 3) | (a >> 29);
	d += ((a & (b | c)) | (b & c)) + nt_buffer[5] + SQRT_2; d = (d << 5) | (d >> 27);
	c += ((d & (a | b)) | (a & b)) + nt_buffer[9] + SQRT_2; c = (c << 9) | (c >> 23);
	b += ((c & (d | a)) | (d & a)) + nt_buffer[13] + SQRT_2; b = (b << 13) | (b >> 19);

	a += ((b & (c | d)) | (c & d)) + nt_buffer[2] + SQRT_2; a = (a << 3) | (a >> 29);
	d += ((a & (b | c)) | (b & c)) + nt_buffer[6] + SQRT_2; d = (d << 5) | (d >> 27);
	c += ((d & (a | b)) | (a & b)) + nt_buffer[10] + SQRT_2; c = (c << 9) | (c >> 23);
	b += ((c & (d | a)) | (d & a)) + nt_buffer[14] + SQRT_2; b = (b << 13) | (b >> 19);

	a += ((b & (c | d)) | (c & d)) + nt_buffer[3] + SQRT_2; a = (a << 3) | (a >> 29);
	d += ((a & (b | c)) | (b & c)) + nt_buffer[7] + SQRT_2; d = (d << 5) | (d >> 27);
	c += ((d & (a | b)) | (a & b)) + nt_buffer[11] + SQRT_2; c = (c << 9) | (c >> 23);
	b += ((c & (d | a)) | (d & a)) + nt_buffer[15] + SQRT_2; b = (b << 13) | (b >> 19);

	/* Round 3 */
	a += (d ^ c ^ b) + nt_buffer[0] + SQRT_3; a = (a << 3) | (a >> 29);
	d += (c ^ b ^ a) + nt_buffer[8] + SQRT_3; d = (d << 9) | (d >> 23);
	c += (b ^ a ^ d) + nt_buffer[4] + SQRT_3; c = (c << 11) | (c >> 21);
	b += (a ^ d ^ c) + nt_buffer[12] + SQRT_3; b = (b << 15) | (b >> 17);

	a += (d ^ c ^ b) + nt_buffer[2] + SQRT_3; a = (a << 3) | (a >> 29);
	d += (c ^ b ^ a) + nt_buffer[10] + SQRT_3; d = (d << 9) | (d >> 23);
	c += (b ^ a ^ d) + nt_buffer[6] + SQRT_3; c = (c << 11) | (c >> 21);
	b += (a ^ d ^ c) + nt_buffer[14] + SQRT_3; b = (b << 15) | (b >> 17);

	a += (d ^ c ^ b) + nt_buffer[1] + SQRT_3; a = (a << 3) | (a >> 29);
	d += (c ^ b ^ a) + nt_buffer[9] + SQRT_3; d = (d << 9) | (d >> 23);
	c += (b ^ a ^ d) + nt_buffer[5] + SQRT_3; c = (c << 11) | (c >> 21);
	b += (a ^ d ^ c) + nt_buffer[13] + SQRT_3; b = (b << 15) | (b >> 17);

	a += (d ^ c ^ b) + nt_buffer[3] + SQRT_3; a = (a << 3) | (a >> 29);
	d += (c ^ b ^ a) + nt_buffer[11] + SQRT_3; d = (d << 9) | (d >> 23);
	c += (b ^ a ^ d) + nt_buffer[7] + SQRT_3; c = (c << 11) | (c >> 21);
	b += (a ^ d ^ c) + nt_buffer[15] + SQRT_3; b = (b << 15) | (b >> 17);

	output[0] = a + INIT_A;
	output[1] = b + INIT_B;
	output[2] = c + INIT_C;
	output[3] = d + INIT_D;
}

__global__ void KernelMain()
{
	int tid = threadIdx.x + (blockIdx.x * blockDim.x);
	int randPasswordLength = tid % maxPasswordLength + 1;	
	char * randomString = new char[randPasswordLength + 1];
	uint32_t buffer[4] = { 0 };
	unsigned char hashOut[16] = { 0 };	

	randomString[randPasswordLength] = 0;
	for (int x = 0; x < randPasswordLength; x++)	
		randomString[x] = validChars[randomBytes[tid + x] % sizeof(validChars)];	

	// Buffer comes out in 4 8-byte parts, reversed.
	NTHash(randomString, randPasswordLength, buffer);
	delete randomString;

	hashOut[3]  = (buffer[0] & 0xFF000000UL) >> 24;
	hashOut[2]  = (buffer[0] & 0x00FF0000UL) >> 16;
	hashOut[1]  = (buffer[0] & 0x0000FF00UL) >> 8;
	hashOut[0]  = (buffer[0] & 0x000000FFUL);

	hashOut[7]  = (buffer[1] & 0xFF000000UL) >> 24;
	hashOut[6]  = (buffer[1] & 0x00FF0000UL) >> 16;
	hashOut[5]  = (buffer[1] & 0x0000FF00UL) >> 8;
	hashOut[4]  = (buffer[1] & 0x000000FFUL);

	hashOut[11] = (buffer[2] & 0xFF000000UL) >> 24;
	hashOut[10] = (buffer[2] & 0x00FF0000UL) >> 16;
	hashOut[9]  = (buffer[2] & 0x0000FF00UL) >> 8;
	hashOut[8]  = (buffer[2] & 0x000000FFUL);

	hashOut[15] = (buffer[3] & 0xFF000000UL) >> 24;
	hashOut[14] = (buffer[3] & 0x00FF0000UL) >> 16;
	hashOut[13] = (buffer[3] & 0x0000FF00UL) >> 8;
	hashOut[12] = (buffer[3] & 0x000000FFUL);

	bool match = true;
	for (int x = 0; x < 16; x++)
	{
		if (hashOut[x] != inputHashBytes[x])
		{
			match = false;
			break;
		}
	}

	if (!device_shouldStop)
	if (match)
	{
		device_shouldStop = true;
		printf("\nMatch Found on Thread %d!\n\n%02x%02x%02x%02x%02x%02x%02x%02x%02x%02x%02x%02x%02x%02x%02x%02x = %s\n\n", tid, hashOut[0], hashOut[1],   hashOut[2],  hashOut[3], 
			                                                                                                                    hashOut[4], hashOut[5],   hashOut[6],  hashOut[7],
																								                                hashOut[8], hashOut[9],   hashOut[10], hashOut[11],
																								                                hashOut[12], hashOut[13], hashOut[14], hashOut[15], randomString);
		
	}	
}

// This is the "press any key to quit" interrupt thread.
DWORD WINAPI Interrupt(LPVOID lpParam)
{
	_getch();	
	host_shouldStop = true;	
	return 0;
}

int main(int argc, char **argv)
{
	unsigned char dummyArray[16] = { 0 };
	hipDevice_t device;
	int deviceCount = 0;
	char deviceName[256];
	hipError_t error_id;
	size_t totalGlobalMem;
	int multiProcessorCount = 0;
	int majorCapabilityVersion = 0, minorCapabilityVersion = 0;
	int clockRate = 0, memoryClock = 0, memBusWidth = 0;
	int maxThreadsPerMultiProcessor = 0, maxThreadsPerBlock = 0;
	int maxBlockDim[3], maxGridDim[3];	
	unsigned __int64 startingTime = 0, endingTime = 0, hostTimerfrequency = 0;
	unsigned __int64 hashesGenerated = 0;
	

	printf("\n");

	// Wrong number of command line args, exit.
	if (argc < 2 || argc > 4)
		PrintHelpText("Must supply at least 1 command line argument and no more than 3!");	

	// Loop through all args, convert to lowercase, and validate for correctness.
	for (int x = 1; x < argc; x++)
	{
		int equalsSignsInArg = 0;
		for (int c = 0; c < strlen(argv[x]); c++)
		{
			argv[x][c] = tolower(argv[x][c]);
			if (argv[x][c] == '=')
				equalsSignsInArg++;

		}
		if (equalsSignsInArg != 1)		
			PrintHelpText("Argument syntax error - number of equals signs was not 1!");		

		// hash= is the only mandatory arg that the user must supply.
		if (startsWith(argv[x], "hash="))
		{
			// Everything is offset by 5 because we are skipping the first 5 bytes (hash=).
			if (strlen(argv[x]) != 37)
				PrintHelpText("The supplied hash has incorrect syntax!(Invalid length?)");
			
			// Room for 32 characters plus a null terminator.
			char hashStr[33] = { '\0' };
			for (int v = 5; v < 37; v++)			
				hashStr[v - 5] = argv[x][v];			

			if (!IsMD4HashString(hashStr))			
				PrintHelpText("The supplied hash has incorrect syntax! (Invalid chars?)");			

			// Now convert the hex string into a byte array by grabing two characters at a time and converting them to base 16 numbers.
			for (int j = 0; j < 32; j += 2)
			{
				char b[2] = { 0, 0 };
				b[0] = (char)hashStr[j];
				b[1] = (char)hashStr[j + 1];
				inputHashBytes[j / 2] = (char)strtoul(b, NULL, 16);
			}
			printf("  Searching for Hash: %s\n", hashStr);
		}
		// This command line argument is optional. Everything is offset by 7 characters because we are skipping the first 7 bytes (maxlen=).
		else if (startsWith(argv[x], "maxlen="))
		{
			char maxLenArgBuf[8];
			for (int v = 7; v < 15; v++)			
				maxLenArgBuf[v - 7] = argv[x][v];
			
			maxPasswordLength = strtoul(maxLenArgBuf, NULL, 10);
			if (maxPasswordLength < 1 || maxPasswordLength > 120)			
				PrintHelpText("Max Password Length (maxlen) was out of range!");

		}
		// This command line argument is optional.
		else if (startsWith(argv[x], "preferreddevice="))
		{
			char preferredDeviceBuf[8];
			for (int v = 16; v < 20; v++)			
				preferredDeviceBuf[v - 16] = argv[x][v];
			
			preferredDeviceNum = strtoul(preferredDeviceBuf, NULL, 10);
			if (preferredDeviceNum < 0 || preferredDeviceNum > 1024)			
				PrintHelpText("Preferred Device was out of range!");

		}		
		else
		{
			PrintHelpText("Unrecognized command line argument!");
		}
	}	

	printf(" Max Password Length: %i\n", maxPasswordLength);

	// User apparently did not supply a hash, cannot continue.
	if (memcmp(inputHashBytes, dummyArray, 16) == 0)	
		PrintHelpText("Hash is missing!");

	printf("\n");

	// Must initialize the CUDA driver API first.
	error_id = hipInit(0);
	if (error_id != hipSuccess)
	{
		printf("\nERROR: NVIDIA CUDA initialization returned %s!\n", getCudaDrvErrorString(error_id));
		return 1;
	}
	error_id = hipGetDeviceCount(&deviceCount);
	if (error_id != hipSuccess)
	{
		printf("\nERROR: hipGetDeviceCount reutnred %s!\n", getCudaDrvErrorString(error_id));
		return 1;
	}

	printf("  CUDA Devices Found: %d\n", deviceCount);
	if (deviceCount < 1)
		return 1;
	
	if (preferredDeviceNum > deviceCount - 1)
	{
		printf("\nERROR: The specified preferred device number (%d) was not found!\n", preferredDeviceNum);
		return 1;
	}

	device = preferredDeviceNum;

	// Collect more statistics about the chosen CUDA device.
	error_id = hipDeviceGetName(deviceName, 256, device);
	if (error_id != hipSuccess)
	{
		printf("\nERROR: hipDeviceGetName returned %s!\n", getCudaDrvErrorString(error_id));
		return 1;
	}

	error_id = hipDeviceComputeCapability(&majorCapabilityVersion, &minorCapabilityVersion, device);
	if (error_id != hipSuccess)
	{
		printf("ERROR: hipDeviceComputeCapability returned %s!\n", getCudaDrvErrorString(error_id));
		return 1;
	}
	
	error_id = hipDeviceTotalMem(&totalGlobalMem, device);
	if (error_id != hipSuccess)
	{
		printf("\nERROR: hipDeviceTotalMem returned %s!\n", getCudaDrvErrorString(error_id));
		return 1;
	}
	getCudaAttribute<int>(&multiProcessorCount, hipDeviceAttributeMultiprocessorCount, device);
	getCudaAttribute<int>(&clockRate, hipDeviceAttributeClockRate, device);
	getCudaAttribute<int>(&memoryClock, hipDeviceAttributeMemoryClockRate, device);
	getCudaAttribute<int>(&memBusWidth, hipDeviceAttributeMemoryBusWidth, device);
	getCudaAttribute<int>(&maxThreadsPerMultiProcessor, hipDeviceAttributeMaxThreadsPerMultiProcessor, device);
	getCudaAttribute<int>(&maxThreadsPerBlock, hipDeviceAttributeMaxThreadsPerBlock, device);
	getCudaAttribute<int>(&maxBlockDim[0], hipDeviceAttributeMaxBlockDimX, device);
	getCudaAttribute<int>(&maxBlockDim[1], hipDeviceAttributeMaxBlockDimY, device);
	getCudaAttribute<int>(&maxBlockDim[2], hipDeviceAttributeMaxBlockDimZ, device);
	getCudaAttribute<int>(&maxGridDim[0], hipDeviceAttributeMaxGridDimX, device);
	getCudaAttribute<int>(&maxGridDim[1], hipDeviceAttributeMaxGridDimY, device);
	getCudaAttribute<int>(&maxGridDim[2], hipDeviceAttributeMaxGridDimZ, device);	

	// Display the rest of the specs on the chosen CUDA device.
	printf("Selected Device Name: %s\n", deviceName);
	printf("  Capability Version: %d.%d\n", majorCapabilityVersion, minorCapabilityVersion);
	printf("       Device Memory: %.0f MB\n", (float)totalGlobalMem / 1048576.0f);
	printf("     Multiprocessors: %d\n", multiProcessorCount);
	printf("       CUDA Cores/MP: %d\n", _ConvertSMVer2CoresDRV(majorCapabilityVersion, minorCapabilityVersion));
	printf("    Total CUDA Cores: %d\n", _ConvertSMVer2CoresDRV(majorCapabilityVersion, minorCapabilityVersion) * multiProcessorCount);
	printf("      GPU Clock Rate: %.0f MHz\n", clockRate * 1e-3f);
	printf("   Memory Clock Rate: %.0f MHz\n", memoryClock * 1e-3f);
	printf("    Memory Bus Width: %d bit\n", memBusWidth);
	printf("      Max Threads/MP: %d\n", maxThreadsPerMultiProcessor);
	printf("   Max Threads/Block: %d\n", maxThreadsPerBlock);
	printf("Max Thread Block Dim: x=%d,y=%d,z=%d\n", maxBlockDim[0], maxBlockDim[1], maxBlockDim[2]);
	printf(" Max Grid Dimensions: x=%d,y=%d,z=%d\n", maxGridDim[0],   maxGridDim[1], maxGridDim[2]);

	// Setting up our high precision timer.
	QueryPerformanceFrequency((LARGE_INTEGER *)&hostTimerfrequency);
	if (hostTimerfrequency < 1)
	{
		printf("\nERROR: Unable to query performance frequency!\n");
		return 1;
	}
	printf("\nHashing will now commence. Press any key to interrupt the program.\n");
	QueryPerformanceCounter((LARGE_INTEGER *)&startingTime);
	// Start the "Press any key to interrupt... thread"
	CreateThread(NULL, 0, Interrupt, NULL, 0, NULL);
	
	dim3 gridDimensions(64, 64, 1);	// x * y * z = the number of blocks being launched, but z must be 1 for compute capability 1.x devices.
	dim3 blockDimensions(8, 8, 8);	// x * y * z = the number of threads per block. Probably should not exceed maxThreadsPerBlock here.

	while (true)
	{	
		if (host_shouldStop || device_shouldStop)
			break;

		// Must generate some GOOD random bytes.		
		if (BCryptGenRandom(NULL, (PBYTE)randomBytes, sizeof(randomBytes), BCRYPT_USE_SYSTEM_PREFERRED_RNG) != 0)
		{
			printf("\nERROR: BCryptGenRandom Error!\n");
			break;
		}
		
		KernelMain <<< gridDimensions, blockDimensions >>>();
		if (hipDeviceSynchronize() != hipSuccess)
		{
			printf("\nERROR: CUDA Kernel Error!\n");
			break;			
		}
		
		hashesGenerated += ((gridDimensions.x * gridDimensions.y * gridDimensions.z) * (blockDimensions.x * blockDimensions.y * blockDimensions.z));
		//printf("%lu\n", hashesGenerated);
	}				
	
	QueryPerformanceCounter((LARGE_INTEGER *)&endingTime);

	hipDeviceReset();
	
	double totalTimeElapsed = (double)(endingTime - startingTime) / hostTimerfrequency;
	printf("\n%lu hashes generated in %.2f seconds.\n", hashesGenerated, totalTimeElapsed);
	printf("%.0f hashes/second.\n", hashesGenerated / totalTimeElapsed);
    
	return 0;
}